
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <vector>

#include <hipfft/hipfft.h>
//#include <cutil_inline.h>
//#include <shrQATest.h>
void fft(std::vector<double> x, std::vector<double> y);

// The filter size is assumed to be a number smaller than the signal size
#define SIGNAL_SIZE       1024
#define cutilSafeCall(x) x
#define cufftSafeCall(x) x
int main(int argc, char** argv)
{
    std::vector<double> x(SIGNAL_SIZE), y(SIGNAL_SIZE);
    for (unsigned int i = 0; i < SIGNAL_SIZE; ++i) {
        x[i] = rand() / (double)RAND_MAX;
        y[i] = 0;
    }
    
    fft(x,y);
  

}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void fft(std::vector<double> x, std::vector<double> y)
{
    uint SIZE = x.size();
    
    printf("[1DCUFFT] is starting...\n");

    hipfftComplex* h_signal=(hipfftComplex*)malloc(sizeof(hipfftComplex) * SIZE);
    // Allocate host memory for the signal
    //Complex* h_signal = (Complex*)malloc(sizeof(Complex) * SIZE);
    // Initalize the memory for the signal
    for (unsigned int i = 0; i < SIZE; ++i) {
        h_signal[i].x = x[i];
        h_signal[i].y = y[i];
    }

    int mem_size = sizeof(hipfftComplex) * SIZE;

    // Allocate device memory for signal
    hipfftComplex* d_signal;
    cutilSafeCall(hipMalloc((void**)&d_signal, mem_size));

    // Copy host memory to device
    cutilSafeCall(hipMemcpy(d_signal, h_signal, mem_size,
                              hipMemcpyHostToDevice));



    // CUFFT plan
    hipfftHandle plan;
    cufftSafeCall(hipfftPlan1d(&plan, SIZE, HIPFFT_C2C, 1));

    // Transform signal
    printf("Transforming signal cufftExecC2C\n");
    cufftSafeCall(hipfftExecC2C(plan, (hipfftComplex *)d_signal, (hipfftComplex *)d_signal, HIPFFT_FORWARD));

    // Transform signal back
    printf("Transforming signal back cufftExecC2C\n");
    cufftSafeCall(hipfftExecC2C(plan, (hipfftComplex *)d_signal, (hipfftComplex *)d_signal, HIPFFT_BACKWARD));

    // Copy device memory to host
    hipfftComplex* h_inverse_signal = (hipfftComplex*)malloc(sizeof(hipfftComplex) * SIZE);
    cutilSafeCall(hipMemcpy(h_inverse_signal, d_signal, mem_size, hipMemcpyDeviceToHost));

    for(int i=0;i< SIZE;i++){
        h_inverse_signal[i].x= h_inverse_signal[i].x/(float)SIZE;
        h_inverse_signal[i].y= h_inverse_signal[i].y/(float)SIZE;

        printf("Residule : %f %f\n",h_signal[i].x-h_inverse_signal[i].x, h_signal[i].y-h_inverse_signal[i].y);
    }  



    //Destroy CUFFT context
    cufftSafeCall(hipfftDestroy(plan));

    // cleanup memory
    free(h_signal);

    free(h_inverse_signal);
    hipFree(d_signal);
}
